#include <cyclic-reduction/cu_cr_solver.h>
#include <cyclic-reduction/cu_cr_internal.h>
#include <cyclic-reduction/cu_cr_functors.cu>

#include <hip/hip_runtime.h>
#include <math.h>
#include <thread>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/system/cuda/execution_policy.h>

/*
* For method documentation see cu_cr_internal.h unless otherwise specified.
*/

namespace cyclic_reduction{

HVectorD Solve(int size, HVectorD h_vect_a, HVectorD h_vect_b, HVectorD h_vect_c, HVectorD h_vect_d){

	DVectorD d_vect_a,
		d_vect_b,
		d_vect_c,
		d_vect_d,
		d_vect_x(size,0.00),
		d_vect_a_prime(size,0.00),
		d_vect_c_prime(size,0.00),
		d_vect_temp(size,0.00);

	d_vect_a = h_vect_a;
	d_vect_b = h_vect_b;
	d_vect_c = h_vect_c;
	d_vect_d = h_vect_d;

//Define and create Cuda Streams
	
	hipStream_t s1,s2,s3;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	hipStreamCreate(&s3);

//Foward Reduction Phase

	int level = 1;
	while(level < size){

	//AlphaBeta Methods
		d_vect_x = d_vect_d;

		LowerAlphaBeta(size,level,
			d_vect_a.data(),
			d_vect_a_prime.data(),
			d_vect_b.data()
		);

		UpperAlphaBeta(size, level,
			d_vect_b.data(),
			d_vect_c.data(),
			d_vect_c_prime.data()
		);
	
	
	//Front Methods
		
		MainFront(size, level,
			d_vect_a_prime.data(),
			d_vect_b.data(),
			d_vect_c.data(),
			d_vect_temp.data()
		);

		SolutionFront(size, level,
			d_vect_a_prime.data(),
			d_vect_d.data(),
			d_vect_x.data(),
			d_vect_temp.data()
		);

		LowerFront(size, level,
			d_vect_a.data(),
			d_vect_a_prime.data()
		);

	//Back Methods

		MainBack(size, level,
			d_vect_a.data(),
			d_vect_c_prime.data(),
			d_vect_b.data(),
			d_vect_temp.data()
		);

		SolutionBack(size, level,
			d_vect_c_prime.data(),
			d_vect_d.data(),
			d_vect_x.data(),
			d_vect_temp.data()
		);

		UpperBack(size, level,
			d_vect_c.data(),
			d_vect_c_prime.data()
		);			

	//Set up diagonals for next reduction level
		d_vect_a = d_vect_a_prime;
		d_vect_c = d_vect_c_prime;
		d_vect_d = d_vect_x;

		level *= 2;
	}

//Backward Substitution Phase
	DVectorD d_vect_results(size);
		thrust::transform(
			d_vect_d.begin(), d_vect_d.end(),
			d_vect_b.begin(),
			d_vect_results.begin(),
			thrust::divides<double>()
		);

	h_vect_d = d_vect_results;

	hipStreamDestroy(s1);
	hipStreamDestroy(s2);
	hipStreamDestroy(s3);	
				
	return h_vect_d;
}


void LowerAlphaBeta(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime, DPtrD d_ptr_b){

	thrust::transform(
		d_ptr_a + level, d_ptr_a + n,
		d_ptr_b,
		d_ptr_a_prime + level,
		AlphaBeta()
	);
		
}

void UpperAlphaBeta(int n, int level, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	thrust::transform(
		d_ptr_c , d_ptr_c + (n-level),
		d_ptr_b + level,
		d_ptr_c_prime,
		AlphaBeta()
	);

}

void MainFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_b, DPtrD d_ptr_c, DPtrD d_ptr_temp){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_c,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b + level, d_ptr_b + n,
		d_ptr_temp,
		d_ptr_b + level,
		thrust::plus<double>()
	);

}

void SolutionFront(int n, int level, DPtrD d_ptr_a_prime, DPtrD d_ptr_d, DPtrD d_ptr_x, DPtrD d_ptr_temp ){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_d,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x + level, d_ptr_x + n,
		d_ptr_temp,
		d_ptr_x + level,
		thrust::plus<double>()
	);

}

void LowerFront(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_a_prime){

	thrust::transform(
		d_ptr_a_prime + level, d_ptr_a_prime + n,
		d_ptr_a,
		d_ptr_a_prime + level,
		thrust::multiplies<double>()
	);	
}

void MainBack(int n, int level, DPtrD d_ptr_a, DPtrD d_ptr_c_prime, DPtrD d_ptr_b, DPtrD d_ptr_temp){

	thrust::transform(
		d_ptr_c_prime , d_ptr_c_prime + (n - level),
		d_ptr_a + level,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_b , d_ptr_b + (n - level),
		d_ptr_temp,
		d_ptr_b,
		thrust::plus<double>()
	);
}

void SolutionBack(int n, int level, DPtrD d_ptr_c_prime, DPtrD d_ptr_d, DPtrD d_ptr_x, DPtrD d_ptr_temp){
	
	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_d + level,
		d_ptr_temp,
		thrust::multiplies<double>()
	);

	thrust::transform(
		d_ptr_x , d_ptr_x + (n-level),
		d_ptr_temp,
		d_ptr_x,
		thrust::plus<double>()
	);

}


void UpperBack(int n, int level, DPtrD d_ptr_c, DPtrD d_ptr_c_prime){

	thrust::transform(
		d_ptr_c_prime, d_ptr_c_prime + (n-level),
		d_ptr_c + level,
		d_ptr_c_prime,
		thrust::multiplies<double>()
	);	
}

}//END - namespace
